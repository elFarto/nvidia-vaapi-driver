
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void convert_nv12_bt701_block_linear(uint8_t *out, uint8_t *luma, uint8_t *chroma, uint32_t width, uint32_t height, uint32_t log2GobsPerBlockX, uint32_t log2GobsPerBlockY) {

    //TODO these need to be passed in
    uint32_t gobWidth    = 16;//px TODO calculate these from hardware
    uint32_t gobHeight   = 8;//px
    uint32_t bytesPerPixel = 4;//bpc * channels / 8;

//    uint32_t blockWidth  = gobWidth * (1<<log2GobsPerBlockX);//px
//    uint32_t blockHeight = gobHeight * (1<<log2GobsPerBlockY);//px

    uint32_t gobSize     = gobWidth * gobHeight * bytesPerPixel;
    uint32_t gobsPerX    = gridDim.x;
//    uint32_t gobsPerY    = gridDim.y;

    uint32_t gobX        = blockIdx.x;
    uint32_t gobY        = blockIdx.y;

    uint32_t gobPixelX   = gobX * gobWidth;
    uint32_t gobPixelY   = gobY * gobHeight;

    uint32_t blockY      = gobY / (1<<log2GobsPerBlockY);

    gobY = gobY % (1<<log2GobsPerBlockY);

    uint32_t blockOffset = (blockY * (gobsPerX * (1<<log2GobsPerBlockY) * gobSize));
    uint32_t gobOffset = ((gobX * (1<<log2GobsPerBlockY)) + gobY) * gobSize;

    uint32_t subGobWidth  = 4;//px
    uint32_t subGobHeight = 4;//px

    for (uint32_t i = 0; i < gobSize; i+=4) {
        uint32_t t = i / bytesPerPixel;
        uint32_t idx = (i / 64);
        uint32_t half = idx / 4;

        uint32_t subGobX = idx&1;
        uint32_t subGobY = (idx&2)>>1;

        uint32_t subSubGobX = t&3;
        uint32_t subSubGobY = (t%16)/4;

        uint32_t x = gobPixelX + (half * subGobWidth * 2) + (subGobX * subGobWidth) + subSubGobX;
        uint32_t y = gobPixelY + (subGobY * subGobHeight) + subSubGobY;

        uint32_t pixelOffset = i;
        uint8_t *pixelOut =  out + blockOffset + gobOffset + pixelOffset;

//        pixelOut[2] = t;//R
//        pixelOut[1] = idx;//G
//        pixelOut[0] = 0;//B

        uint8_t *lumaOffset   = luma   + (y*width) + x;
        uint8_t *chromaOffset = chroma + (y>>1)*width + (x & ~1);

        uint8_t Y = lumaOffset[0];
        uint8_t U = chromaOffset[0];
        uint8_t V = chromaOffset[1];

        pixelOut[2] = (uint8_t) fmaxf(fminf(Y + 1.402 * (V - 128), 255.0), 0.0);
        pixelOut[1] = (uint8_t) fmaxf(fminf(Y - 0.34413 * (U - 128) - 0.71414*(V - 128), 255.0), 0.0);
        pixelOut[0] = (uint8_t) fmaxf(fminf(Y + 1.772*(U - 128), 255.0), 0.0);
    }
}